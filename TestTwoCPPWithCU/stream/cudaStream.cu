#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void addWithCuda(int *c, const int *a, const int *b, size_t size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x;
    c[i] = a[i] + b[i];
}
int maindddd()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    addWithCuda(c, a, b, arraySize);
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",c[0],c[1],c[2],c[3],c[4]);
    hipDeviceReset();

    return 0;
}


void addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;

    hipSetDevice(0);
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipStream_t stream[5];
    for(int i = 0;i<5;i++)
    {
        hipStreamCreate(&stream[i]);
    }
    for(int i = 0;i<5;i++)
        addKernel<<<1,1,0,stream[i]>>>(dev_c+i, dev_a+i, dev_b+i);

    hipDeviceSynchronize();

    hipDeviceSynchronize();
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0;i<5;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

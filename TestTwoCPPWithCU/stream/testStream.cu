/*
 * testStream.cu
 *
 *  Created on: Sep 17, 2017
 *      Author: zy
 */

#include <iostream>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

void testDeviceGPU()
{
	hipDeviceProp_t prop;
	int whichdevice;
	hipGetDevice(&whichdevice);
	hipGetDeviceProperties(&prop,whichdevice);
	cout<<prop.deviceOverlap<<endl;
	if(prop.deviceOverlap)
		cout<<"This GPU Support Device Overlap"<<endl;
}

int main234()
{
	cout<<"Begin：    "<<endl;

	cout<<"Game Over"<<endl;
	return 1;
}




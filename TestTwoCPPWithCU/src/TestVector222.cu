#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <vector>
#include <list>
#include <stdio.h>
#include "hello.cuh"

#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
using namespace std;


class Particle
{
public:
    int particle_id;
    float particle_velocity;
    float particle_density;
};

__global__ void kernel_function(Particle *pointer, int total)
{
    int index=blockIdx.x;

    if(index < total)
    {
        pointer[index].particle_id=pointer[index].particle_id+2;
        pointer[index].particle_velocity=pointer[index].particle_velocity*2;
        pointer[index].particle_density=pointer[index].particle_density*3;
    }
}

int mainForClass()
{
	cout<<endl<<"Main FOR CLASS  AAAAAAAAA      "<<endl;
    std::list<Particle> particle_list;

    Particle particle_a;
    Particle particle_b;
    Particle particle_c;

    particle_a.particle_id=1;
    particle_a.particle_velocity=1.1;
    particle_a.particle_density=1.22;

    particle_b.particle_id=2;
    particle_b.particle_velocity=2.1;
    particle_b.particle_density=2.22;

    particle_c.particle_id=3;
    particle_c.particle_velocity=3.1;
    particle_c.particle_density=3.22;

    particle_list.push_back(particle_a);
    particle_list.push_back(particle_b);
    particle_list.push_back(particle_c);

    thrust::device_vector<Particle> device_list(particle_list.begin(), particle_list.end());

    Particle *raw_pointer;
    raw_pointer=thrust::raw_pointer_cast(&device_list[0]);

    kernel_function<<<3, 1>>>(raw_pointer, device_list.size());

    thrust::host_vector<Particle> host_list=device_list;

    for(int count=0; count<host_list.size(); count++)
    {
        printf("%d, %f, %f\n", host_list[count].particle_id, host_list[count].particle_velocity, host_list[count].particle_density);
    }

    return 0;
}



int mainForFFFF()
{

	cout<<endl<<"FFFFFFFFFFFFFUCK    "<<endl;
	// initialize all ten integers of a device_vector to 1
	thrust :: device_vector <int > D(10 , 1);

	// set the first seven elements of a vector to 9
	thrust :: fill (D. begin () , D. begin () + 7, 9);

	// initialize a host_vector with the first five elements of D
	thrust :: host_vector <int > H(D. begin () , D. begin () + 5);

	// set the elements of H to 0, 1, 2, 3, ...
	thrust :: sequence (H. begin () , H. end ());

	// copy all of H back to the beginning of D
	thrust :: copy (H. begin () , H. end () , D. begin ());
	// print D
	for ( int i = 0; i < D. size (); i ++)
		std::cout << "D[" << i << "] = " << D[i] <<"  ";
	cout<<endl;
	return 0;
}

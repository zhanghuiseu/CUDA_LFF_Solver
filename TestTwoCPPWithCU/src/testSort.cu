#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
using namespace std;

#include "hip/hip_runtime.h"
#include ""

#include "hello.cuh"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

int mainForSort()
{
	const int SIZE=1000000;


	cout<<"Begin :   "<<endl;
	srand(time(0));
	double* a = new double[SIZE];
	double* b = new double[SIZE];
	for (long i = 0; i < SIZE; i++)
		b[i] = a[i] = rand()%100;

	double* dev_a=0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_a, sizeof(double)*SIZE);
	hipMemcpy(dev_a,a,SIZE * sizeof(double),hipMemcpyHostToDevice);


	clock_t start, finish;
	start = clock();
    sort(a,a+SIZE);
	finish = clock();
	cout << "By CPU STL Sort: " << (double)(finish - start) / CLOCKS_PER_SEC << endl;


	start = clock();
	thrust::sort(b,b+SIZE);
	finish = clock();
	cout << "By CUDA Thrust Sort: " << (double)(finish - start)/CLOCKS_PER_SEC << endl;

	hipFree(dev_a);
	return 1;
}





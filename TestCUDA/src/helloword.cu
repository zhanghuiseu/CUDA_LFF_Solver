#include <iostream>
#include <cstdlib>
#include <cmath>
#include <ctime>
using namespace std;

#include "hip/hip_runtime.h"


static const long long SIZE = 4096*4096*4;

hipError_t calaAddArray(const double* a, const double* b, double* res, const long SIZE);

__global__ void addArray(const double* a,const double *b, double* c)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	while (i < SIZE)
	{
		c[i] = (a[i] * b[i]-b[i]*3.1415926)/a[i] * b[i]*b[i]-a[i]*b[i];
		i += gridDim.x*blockDim.x;;
	}
}

int main4444()
{
	cout<<"Begin :   "<<endl;
	srand(time(0));
	double* a = new double[SIZE];
	double* b = new double[SIZE];
	for (long i = 0; i < SIZE; i++)
	{
		a[i] = (double)rand() * (double)rand();
		b[i] = (double)rand() * (double)rand();
	}
	double* res = new double[SIZE];
	

	int count=2;
	clock_t start, finish;
	start = clock();
	for(int i=0;i<count;i++)
	{
		for (long long i = 0; i < SIZE; i++)
			res[i] = (a[i] * b[i]-b[i]*3.1415926)/a[i] * b[i]*b[i]-a[i]*b[i];
	}

	finish = clock();
	/*
	for (int i = 0; i < SIZE; i++)
		cout << a[i] << ", ";
	cout << endl;
	for (int i = 0; i < SIZE; i++)
		cout << b[i] << ", ";
	cout << endl;
	for (int i = 0; i < SIZE; i++)
		cout << res[i] << ", ";
	cout << endl << endl << endl;
	*/
	for (long long i = 0; i < SIZE; i++)
	{
		if (res[i] != (a[i] * b[i]-b[i]*3.1415926)/a[i] * b[i]*b[i]-a[i]*b[i] )
		{
			cout << "***********  Failed CPU" << endl;
			break;
		}
	}
	cout << start << "   " << finish << endl;
	cout << "By CPU Array Mult: " << (double)(finish - start) / CLOCKS_PER_SEC << endl;
	
	start = clock();
	for(int i=0;i<count;i++)
	{
		hipError_t cudaErrorRes = calaAddArray(a, b, res, SIZE);
		if (cudaErrorRes != hipSuccess)
		{
			cout << "*************   CUDA Return Result Failed " << endl;
		}
	}
	finish = clock();

/*
	for (int i = 0; i < SIZE; i++)
		cout << a[i] << ", ";
	cout << endl;
	for (int i = 0; i < SIZE; i++)
		cout << b[i] << ", ";
	cout << endl;
	*/
	for (long long i = 0; i < SIZE; i++)
	{
		if (res[i] != (a[i] * b[i]-b[i]*3.1415926)/a[i] * b[i]*b[i]-a[i]*b[i])
		{
			cout << "*************   Failed GPU" << endl;
			break;
		}
	}

	cout << endl;
	cout << start << "   " << finish << endl;
	cout << "By CUDA Array Mult: " << (double)(finish - start)/CLOCKS_PER_SEC << endl;

	cout<<"Game Over!!!!"<<endl;
	return 1;
}

hipError_t calaAddArray(const double* a, const double* b, double* c, const long SIZE)
{
	double* dev_a=0;
	double* dev_b=0;
	double* dev_c=0;
	hipError_t cudaStatus=hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		cout << "CUDA ADD Failed In Function hipError_t calaAddArray " << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, sizeof(double)*SIZE);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc decvA failed!" << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, sizeof(double)*SIZE);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc decvB failed!" << endl;
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, sizeof(double)*SIZE);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMalloc decvC failed!" << endl;
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a,a,SIZE * sizeof(double),hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy decvA failed!" << endl;
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, SIZE * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy decvB failed!" << endl;
		goto Error;
	}

	addArray <<<4096,1024>>>(dev_a, dev_b, dev_c);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		cout << "addKernel launch failed: " <<hipGetErrorString(cudaStatus)<<endl;
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		cout<<cudaStatus<<" hipDeviceSynchronize returned error code %d after launching addKernel!"<<endl;
		goto Error;
	}

	cudaStatus = hipMemcpy(c, dev_c, SIZE * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		cout << "hipMemcpy decvC failed!" << endl;
		goto Error;
	}
Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return cudaStatus;
}

#include "hip/hip_runtime.h"
///////////////////////
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
using namespace std;


#if __DEVICE_EMULATION__
bool InitCUDA(void) { return true; }
#else
bool InitCUDA(void)
{
	int count = 0;
	int i = 0;
	hipGetDeviceCount(&count);
	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA.\n");
		return false;
	}
	hipSetDevice(i);
	printf("CUDA initialized.\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, i);
	printf("Device : \" %s \" \n\n", prop.name);
	return true;
}
#endif

#define aW 855+800
#define aH 5110
#define bW 1013+800
#define blocknum 128//32
#define threadnum 512//256

typedef struct
{
	int width;
	int height;
	int *element;
}Matrix;

Matrix InitMatrix(int w, int h)
{
	Matrix t;
	t.element = (int *)malloc(w * h * sizeof(int));
	for (int i = 0; i < w*h; i++)
		t.element[i] = rand() % 10;
	t.width = w;
	t.height = h;
	return t;
}
Matrix MM(Matrix a, Matrix b)
{
	Matrix t;
	t.element = (int *)malloc(a.height * b.width * sizeof(int));
	t.width = b.width;
	t.height = a.height;
	int x;
	int y;
	for (int i = 0; i < t.width * t.height; i++)
	{
		x = i / t.width * a.width;
		y = i - i / t.width * t.width;
		t.element[i] = 0;
		for (int k = 0; k < a.width; k++)
		{
			t.element[i] += a.element[x + k] * b.element[y + b.width * k];
		}
	}
	return t;
}




__global__ static void MatrixMul(int *ma, int *mb, int *mc, int *mp)
{
	int aw = mp[0];
	int bw = mp[2];
	int cw = mp[4];
	int ch = mp[5];
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	int i, x, y;

	for (i = bid * threadnum + tid; i < cw * ch; i += threadnum * blocknum)
	{
		x = i / cw * aw;
		y = i - i / cw * cw;
		mc[i] = 0;
		for (int k = 0; k < aw; k++)
		{
			mc[i] += ma[x + k] * mb[y + k * bw];
		}
	}
}




int main(int argc, char* argv[])
{
	cout << "Matrix Mult Begin : " << endl;
	if (!InitCUDA()) 
	{
		return 0;
	}
	//�������
	//int matrixa[N][N] , matrixb[N][N] , matrixc[N][N] , gpuresult[N][N] , matrixd[N][N] ;
	Matrix matrixa = InitMatrix(aW, aH);
	Matrix matrixb = InitMatrix(bW, aW);
	Matrix matrixc;
	Matrix gpuresult = InitMatrix(bW, aH);

	int matrixprop[6];

	//CPU�������
	int start = clock();
	matrixc = MM(matrixa, matrixb);
	int finish = clock();
	double time = ((double)finish - (double)start) / CLOCKS_PER_SEC;
	cout << "CPU Time is " << time << endl;

	start = clock();
	matrixprop[0] = matrixa.width;
	matrixprop[1] = matrixa.height;
	matrixprop[2] = matrixb.width;
	matrixprop[3] = matrixb.height;
	matrixprop[4] = matrixc.width;
	matrixprop[5] = matrixc.height;

	//�����Դ�
	int *ma, *mb, *mc, *mp;
	hipMalloc((void**)&ma, sizeof(int) * matrixa.width * matrixa.height);
	hipMalloc((void**)&mb, sizeof(int) * matrixb.width * matrixb.height);
	hipMalloc((void**)&mc, sizeof(int) * matrixc.width * matrixc.height);
	hipMalloc((void**)&mp, sizeof(int) * 6);
	//����ݸ��Ƶ��Դ���
	hipMemcpy(ma, matrixa.element, sizeof(int) * matrixa.width * matrixa.height, hipMemcpyHostToDevice);
	hipMemcpy(mb, matrixb.element, sizeof(int) * matrixb.width * matrixb.height, hipMemcpyHostToDevice);
	hipMemcpy(mp, matrixprop, sizeof(int) * 6, hipMemcpyHostToDevice);

	//����CUDA����
	MatrixMul << < blocknum, threadnum>> >(ma, mb, mc, mp);
	hipDeviceSynchronize();
	//����ݴ��Դ��и��Ƴ���
	hipMemcpy(gpuresult.element, mc, sizeof(int) * gpuresult.width * gpuresult.height, hipMemcpyDeviceToHost);
	
	finish = clock();
	time = ((double)finish - (double)start) / CLOCKS_PER_SEC;
	cout << "GPU Time is " << time << endl;


	for (int i = 0; i < gpuresult.width * gpuresult.height; i++)
	{
		//printf("%d -- %d\n",matrixc.element[ i ],gpuresult.element[ i ]);
		if (matrixc.element[i] != gpuresult.element[i])
		{
			printf("ERROR");
		}
	}

	hipFree(ma);
	hipFree(mb);
	hipFree(mc);
	hipFree(mp);

	return 0;
}
